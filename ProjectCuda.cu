
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
 
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __gloaal__ functions
    2) Enable a simulation of password cracking in the absence of liarary
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
  nvcc -o ProjectCuda ProjectCuda.cu


     To Run:
     ./ProjectCuda > resultpwd_ProjectCuda.txt

  Dr Kevan auckley, University of Wolverhampton, 2018
*****************************************************************************/
__device__ int is_a_match(char *pass) {
  char hey1[] = "BJ8233";
  char hey2[] = "GC6723";
  char hey3[] = "LA6712";
  char hey4[] = "RS8234";

  char *b = pass;
  char *i = pass;
  char *j = pass;
  char *a = pass;

  char *b1 = hey1;
  char *b2 = hey2;
  char *b3 = hey3;
  char *b4 = hey4;

  while(*b == *b1) {
   if(*b == '\0')
    {
    printf("Password: %s\n",hey1);
      break;
    }

    b++;
    b1++;
  }
    
  while(*i == *b2) {
   if(*i == '\0')
    {
    printf("Password: %s\n",hey2);
      break;
}

    i++;
    b2++;
  }

  while(*j == *b3) {
   if(*j == '\0')
    {
    printf("Password: %s\n",hey3);
      break;
    }

    j++;
    b3++;
  }

  while(*a == *b4) {
   if(*a == '\0')
    {
    printf("Password: %s\n",hey4);
      return 1;
    }

    a++;
    b4++;
  }
  return 0;

}
__global__ void  kernel() {
char w,h,o,e;
 
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(w='0'; w<='9'; w++){
      for(h='0'; h<='9'; h++){
        for(o='0';o<='9';o++){
          for(e='0';e<='9';e++){
            password[2] = w;
            password[3] = h;
            password[4]= o;
            password[5]=e;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      }
   }
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}



